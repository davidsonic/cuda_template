#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <time.h>

using namespace std;

__global__ void calc_inde(int * hdata, int size){

	int tid = blockDim.x * threadIdx.y + threadIdx.x;
	int num_thread_in_block = blockDim.x * blockDim.y;
	int block_offset = num_thread_in_block * blockIdx.x;

	int num_thread_in_row = num_thread_in_block * gridDim.x;
	int row_offset = num_thread_in_row * blockIdx.y;

	int gid = tid + row_offset + block_offset;

	printf("tid = %d, gid = %d, num = %d\n", tid, gid, hdata[gid]);


}

int main2(){
	int size = 16;
	int byte_size = sizeof(int)*size;
	int *h = (int*)malloc(byte_size);

	time_t time;
	srand((unsigned)time);
	for (int i=0;i<size;i++){
		h[i] = (int)(rand()%0xff);
	}

	int *cudaData;
	hipMalloc((int**)&cudaData, byte_size);
	hipMemcpy(cudaData, h, byte_size, hipMemcpyHostToDevice);

	dim3 grid(2,2);
	dim3 block(2,2);
	calc_inde<< <grid, block>> >(cudaData, size);

	hipDeviceSynchronize();
	hipDeviceReset();


	return 0;
}
